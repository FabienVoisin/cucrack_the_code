#include "hip/hip_runtime.h"
#include <iostream>
#include "gpuutilsfunc.cuh"
#include "testcode.cuh"
#include <stdio.h>
void printcode(player &player){
    
    for (int i =0;i <CODESIZE; i++){
        std::cout<<player.hostcode[i]<<",";
    }
    
    std::cout<<std::endl;
    
}
__host__ void runtestkernel(){
//This function is a test function meant to test whether the constant memory is correctly allocated.
    unsigned int blockspergrid=CODESIZE/THREADSPERBLOCK+1; //define the number
    unsigned int hostresult[CODESIZE];
    unsigned int *deviceresult;
    deviceresult=allocate_device_memory(CODESIZE);
    
    for (int i=0;i<CODESIZE;i++){
        hostresult[i]=0;
    }
    std::cout<<"hello world"<<std::endl;
    testkernel<<<blockspergrid,THREADSPERBLOCK>>>(deviceresult);
    copy_device_to_output(deviceresult,hostresult,CODESIZE);
    for (int i=0;i<CODESIZE;i++){
        std::cout<<hostresult[i]<<",";
    }
    std::cout<<std::endl;

}

__global__ void testkernel(unsigned int *deviceresult){
    // Set up index for the kernel
    int blockId=blockIdx.z*(gridDim.x*gridDim.y)+blockIdx.y*(gridDim.x)+blockIdx.x;
    int index=blockId*(blockDim.x*blockDim.y*blockDim.z)+threadIdx.z*(blockDim.x*blockDim.y)+threadIdx.y*(blockDim.y)+threadIdx.x;
    if (index<CODESIZE){ //The number of threads may exceed the CODESIZE
        //deviceresult[index]=codetocrack1[index]+2;
        deviceresult[index]=2;
    }

}