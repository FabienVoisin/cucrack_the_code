#include "hip/hip_runtime.h"
#include <iostream>
#include "gpuutilsfunc.cuh"
#include "testcode.cuh"
#include <stdio.h>

__constant__  unsigned int codetocrack1[CODESIZE]; //2 is for 2 players
__constant__  unsigned int codetocrack2[CODESIZE];

void printcode(player &player){
    
    for (int i =0;i <CODESIZE; i++){
        std::cout<<player.hostcode[i]<<",";
    }
    
    std::cout<<std::endl;
    
}
__host__ void runtestkernel(unsigned int *playercode){
//This function is a test function meant to test whether the constant memory is correctly allocated.
    unsigned int blockspergrid=CODESIZE/THREADSPERBLOCK+1; //define the number
    unsigned int hostresult[CODESIZE];
    unsigned int *deviceresult;
    deviceresult=allocate_device_memory(CODESIZE);
    size_t size=CODESIZE*sizeof(unsigned int);
    hipError_t err=hipMemcpyToSymbol(HIP_SYMBOL(codetocrack1),playercode,size);
    //std::cout<<"hello world"<<std::endl;
    testkernel<<<blockspergrid,THREADSPERBLOCK>>>(deviceresult);
    copy_device_to_output(deviceresult,hostresult,CODESIZE);
    for (int i=0;i<CODESIZE;i++){
        std::cout<<hostresult[i]<<","<<playercode[i]<<std::endl;
    }
    std::cout<<std::endl;

}

__global__ void testkernel(unsigned int *deviceresult){
    // Set up index for the kernel
    int blockId=blockIdx.z*(gridDim.x*gridDim.y)+blockIdx.y*(gridDim.x)+blockIdx.x;
    int index=blockId*(blockDim.x*blockDim.y*blockDim.z)+threadIdx.z*(blockDim.x*blockDim.y)+threadIdx.y*(blockDim.y)+threadIdx.x;
    //printf("%u",codetocrack1[index]);
    if (index<CODESIZE){ //The number of threads may exceed the CODESIZE
        deviceresult[index]=codetocrack1[index]+100;
        //deviceresult[index]=2;
    }

}