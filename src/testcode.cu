#include "hip/hip_runtime.h"
#include <iostream>
#include "gpuutilsfunc.cuh"
#include "testcode.cuh"
#include <stdio.h>

__constant__  unsigned int codetocrack1[CODESIZE]; //2 is for 2 players
__constant__  unsigned int codetocrack2[CODESIZE];

void printcode(player &player){
    
    for (int i =0;i <CODESIZE; i++){
        std::cout<<player.hostcode[i]<<",";
    }
    
    std::cout<<std::endl;
    
}
__host__ void runtestkernel(unsigned int *playercode1,unsigned int *playercode2){
//This function is a test function meant to test whether the constant memory is correctly allocated.
    unsigned int blockspergrid=CODESIZE/THREADSPERBLOCK+1; //define the number
    unsigned int hostresult[CODESIZE];
    unsigned int *deviceresult;
    deviceresult=allocate_device_memory(CODESIZE);
    size_t size=CODESIZE*sizeof(unsigned int);
    hipError_t err=hipMemcpyToSymbol(HIP_SYMBOL(codetocrack1),playercode1,size);
    err=hipMemcpyToSymbol(HIP_SYMBOL(codetocrack2),playercode2,size);
    //std::cout<<"hello world"<<std::endl;
    testkernel1<<<blockspergrid,THREADSPERBLOCK>>>(deviceresult);
    copy_device_to_output(deviceresult,hostresult,CODESIZE);
    for (int i=0;i<CODESIZE;i++){
        std::cout<<hostresult[i]<<","<<playercode1[i]<<std::endl;
    }
    std::cout<<std::endl;
    std::cout<<"kernel 2" <<std::endl;
    testkernel2<<<blockspergrid,THREADSPERBLOCK>>>(deviceresult);
    copy_device_to_output(deviceresult,hostresult,CODESIZE);
    for (int i=0;i<CODESIZE;i++){
        std::cout<<hostresult[i]<<","<<playercode2[i]<<std::endl;
    }
    
}

__global__ void testkernel1(unsigned int *deviceresult){
    // Set up index for the kernel
    int blockId=blockIdx.z*(gridDim.x*gridDim.y)+blockIdx.y*(gridDim.x)+blockIdx.x;
    int index=blockId*(blockDim.x*blockDim.y*blockDim.z)+threadIdx.z*(blockDim.x*blockDim.y)+threadIdx.y*(blockDim.y)+threadIdx.x;
    //printf("%u",codetocrack1[index]);
    if (index<CODESIZE){ //The number of threads may exceed the CODESIZE
        deviceresult[index]=codetocrack1[index]+100;
        //deviceresult[index]=2;
    }
}
__global__ void testkernel2(unsigned int *deviceresult){
    // Set up index for the kernel
    int blockId=blockIdx.z*(gridDim.x*gridDim.y)+blockIdx.y*(gridDim.x)+blockIdx.x;
    int index=blockId*(blockDim.x*blockDim.y*blockDim.z)+threadIdx.z*(blockDim.x*blockDim.y)+threadIdx.y*(blockDim.y)+threadIdx.x;
    //printf("%u",codetocrack1[index]);
    if (index<CODESIZE){ //The number of threads may exceed the CODESIZE
        deviceresult[index]=codetocrack2[index]+100;
        //deviceresult[index]=2;
    }
}