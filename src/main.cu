#include <iostream>
#include <thread>
#include <vector>
//#include "struct.cuh"
#include "gpuutilsfunc.cuh"
#include "testcode.cuh"
#include <unistd.h>


int main(){
/*First we need to check how many GPU devices this  */
    int numberofGPUdevice;
    hipGetDeviceCount(&numberofGPUdevice);
    if (numberofGPUdevice<2){
        std::cout<<"We currently need two GPUs to play this game";
    }

//Now we set up the player classes, this will vreate a random code that
// we will need to send to symbol on each GPUs.
    player player1(1);
    player player2(4);
    player1.get_gpu(0);
    player1.get_gpu(1);
    
    /* Part of code to test and delete after*/
    printcode(player1);
    std::cout<<"Now for player 2" << std::endl;
    printcode(player2);
    /* Now we need to copy the data to symbol*/
    
    copy_to_constant_gpu_memory(player1,1);
    copy_to_constant_gpu_memory(player2,0);

    unsigned int *devicevalue;
    unsigned int hostvalue;

    devicevalue=allocate_device_memory(1);
    testval<<<8,32>>>(player1.constantmemory,player2.constantmemory,devicevalue);
    copy_device_to_output(devicevalue,&hostvalue,1);
    std::cout<<"val is "<<hostvalue<<std::endl;
    //runtestkernel(player1.hostcode,player2.hos1tcode);
    // We then need to perform a simple operation
}