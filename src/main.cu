#include <iostream>
#include <thread>
#include <vector>
//#include "struct.cuh"
#include "gpuutilsfunc.cuh"
#include "testcode.cuh"


int main(){
/*First we need to check how many GPU devices this  */
    int numberofGPUdevice;
    hipGetDeviceCount(&numberofGPUdevice);
    if (numberofGPUdevice<2){
        std::cout<<"We currently need two GPUs to play this game";
    }

//Now we set up the player classes, this will vreate a random code that
// we will need to send to symbol on each GPUs.
    player player1;
    player player2;
    player1.get_gpu(0);
    player1.get_gpu(1);
    
    /* Part of code to test and delete after*/
    //printcode(player1);
    std::cout<<"Now for player 2" << std::endl;
    //printcode(player2);
    /* Now we need to copy the data to symbol*/
    copy_to_constant_gpu_memory(player2.hostcode,codetocrack1,0);
    //copy_to_constant_gpu_memory(player1.hostcode,codetocrack2,0);
    runtestkernel();
    // We then need to perform a simple operation

}