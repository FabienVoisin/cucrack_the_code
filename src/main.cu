#include <iostream>
#include <thread>
#include <vector>
//#include "struct.cuh"
#include "gpuutilsfunc.cuh"
#include "testcode.cuh"
#include <unistd.h>


int main(){
/*First we need to check how many GPU devices this  */
    int numberofGPUdevice;
    hipGetDeviceCount(&numberofGPUdevice);
    if (numberofGPUdevice<2){
        std::cout<<"We currently need two GPUs to play this game";
    }

//Now we set up the player classes, this will vreate a random code that
// we will need to send to symbol on each GPUs.
    player player1(1);
    player player2(4);
    player1.get_gpu(0);
    player1.get_gpu(1);
    
    /* Part of code to test and delete after*/
    printcode(player1);
    std::cout<<"Now for player 2" << std::endl;
    printcode(player2);
    /* Now we need to copy the data to symbol*/
    
    copy_to_constant_gpu_memory(player1,1);
    copy_to_constant_gpu_memory(player2,0);

    unsigned int *devicevalue;
    unsigned int hostvalue;

    devicevalue=allocate_device_memory<unsigned int>(1);
    //testval<<<8,32>>>(player1.constantmemory,player2.constantmemory,devicevalue);
    copy_device_to_output<unsigned int>(devicevalue,&hostvalue,1);
    //std::cout<<"val is "<<hostvalue<<std::endl;
    getcodeattempt(player1);
    codecheck(player1);
    //runtestkernel(player1.hostcode,player2.hos1tcode);
    // We then need to perform a simple operation
    std::cout<<"player generated code:"<<std::endl;
    for (int i=0; i<CODESIZE;i++){
    std::cout<<player1.currentcodeattempt[i]<<",";
    }
    std::cout<<std::endl;

    std::cout<<"player incorrect indices"<<std::endl;
    std::cout<<"player incorrect numbers: "<< player1.flagincorrectnumber<<std::endl;
    for (int i=0; i<player1.flagincorrectnumber;i++){
        std::cout<<player1.flagincorrect[i]<<",";
        }

    std::cout<<"player swap indices"<<std::endl;
    std::cout<<"player swap numbers: "<< player1.flagswapnumber<<std::endl;
    for (int i=0; i<player1.flagswapnumber;i++){
        std::cout<<player1.flagswap[i]<<",";
        }

}