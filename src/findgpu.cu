#include <findgpu.h>
#include <common.cuh>

int findGPU(player &player){
    int currentchosendevicenumber=-1;
    int ndevices; // number of GPU devices 
    hipGetDeviceCount(&ndevices);
    for (int i; i<ndevices; i++){
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop);
    hipSetDevice(i);
    }
    /*Check whether the GPU has already been taken*/


    
}