#include "hip/hip_runtime.h"
/*This file provides all the information data into constant, pinned memory and GPU memory*/
#include <iostream>
#include "common.cuh"
#include "gpuutilsfunc.cuh"
#include <string.h>
__constant__  unsigned int codetocrack1[CODESIZE]; //2 is for 2 players
__constant__  unsigned int codetocrack2[CODESIZE];

__global__ void testval(unsigned int *constantmemory1, unsigned int *constantmemory2,unsigned int *deviceval){
    int blockId=blockIdx.z*(gridDim.x*gridDim.y)+blockIdx.y*(gridDim.x)+blockIdx.x;
    int index=blockId*(blockDim.x*blockDim.y*blockDim.z)+threadIdx.z*(blockDim.x*blockDim.y)+threadIdx.y*(blockDim.y)+threadIdx.x;
    if (index==0) *deviceval=constantmemory1[0]+constantmemory2[2];
    
        
    }
__host__  void copy_to_constant_gpu_memory(player &player,int playernumber){
    /* a reminder that the constant memory is read only and 64kB wide.
     we may need to make sure whether we should set a function to make sure the total 
     codesize does not exceed this number for performance purposes */
     size_t size=CODESIZE*sizeof(unsigned int);
     if (playernumber==0){
        hipError_t err=hipMemcpyToSymbol(HIP_SYMBOL(codetocrack1),player.hostcode,size);
        hipGetSymbolAddress((void**)&player.constantmemory,codetocrack1);
     }

     else{
        hipError_t err=hipMemcpyToSymbol(HIP_SYMBOL(codetocrack2),player.hostcode,size);
        hipGetSymbolAddress((void**)&player.constantmemory,codetocrack2);
     }
    
        /*if(strcmp(deviceconstantmemory,"crack2")==0){
            std::cout<<"test"<<std::endl;
            hipError_t err=hipMemcpyToSymbol(HIP_SYMBOL(codetocrack1),hostcodedevice,size);
            if (err!=hipSuccess){
                printf("%s",hipGetErrorString(err));
                exit(1);
        }
        
        }*/
       
        std::cout<<"test"<<std::endl;
    //We must make sure that if we end up with one GPU only, the offset gets propagated
}
template <typename T>
__host__ void copy_input_to_device(T *hostdata, T *devicedata, T numelements){
    hipError_t error;
    size_t size=numelements*sizeof(T);
    error=hipMemcpy(devicedata,hostdata,size,hipMemcpyHostToDevice);

}
template <typename T>
__host__ void copy_device_to_output(T *devicedata, T *hostdata, T numelements){
    hipError_t error;
    size_t size=numelements*sizeof(T);
    error=hipMemcpy(hostdata,devicedata,size,hipMemcpyDeviceToHost);
    if(error!=hipSuccess){
        printf("There is an error here %s\n",hipGetErrorName(error));
        exit(1);
    }
}

template <typename T>
__host__ T* allocate_device_memory(unsigned int numelements){
    hipError_t error;
    T *devicedata;
    size_t size=numelements*sizeof(unsigned int);
    error=hipMalloc((void**)&devicedata,size);

    return devicedata;
}


__host__ void codecheck(player &player){
    get_incorrect_swap_array(player);
    //get_swap_array(player);


    /* I need to free all pointers and CUDA variables*/
    
}


__host__ void get_incorrect_swap_array(player &player){
    unsigned int blockspergrid=CODESIZE/THREADSPERBLOCK+1;
    unsigned int *devicecode;
    unsigned int *device_incorrect_number;
    unsigned int *device_incorrect_array;
    unsigned int *device_swap_number;
    unsigned int *device_swap_array;
    int *incorrectcheck; //array of integers to set -1 for correct values
    int *swapcheck;
    unsigned int host_incorrect_number=0;
    
    incorrectcheck=allocate_device_memory<int>(CODESIZE);
    swapcheck=allocate_device_memory<int>(CODESIZE);
    devicecode=allocate_device_memory<unsigned int>(CODESIZE);
    device_incorrect_number=allocate_device_memory<unsigned int>(1);
    device_swap_number=allocate_device_memory<unsigned int>(1);

    copy_input_to_device<unsigned int>(player.currentcodeattempt,devicecode,CODESIZE);
    copy_input_to_device<unsigned int>(&player.flagincorrectnumber,device_incorrect_number,1);
    copy_input_to_device<unsigned int>(&player.flagswapnumber,device_swap_number,1);
    gpudevicecheckswapincorrect<<<blockspergrid,THREADSPERBLOCK>>>(devicecode,player.constantmemory,incorrectcheck,swapcheck,device_incorrect_number,device_swap_number,CODESIZE);

    /*Place the data back into the player values*/
    /*I will need to free these values before reinitialize them*/
    std::cout<<"am I here?"<<std::endl;
    copy_device_to_output<unsigned int>(device_incorrect_number,&player.flagincorrectnumber,1);
    copy_device_to_output<unsigned int>(device_swap_number,&player.flagswapnumber,1);
    device_incorrect_array=allocate_device_memory<unsigned int>(player.flagincorrectnumber);
    device_swap_array=allocate_device_memory<unsigned int>(player.flagswapnumber);

    unsigned int incorrect_num_threads=min(32,player.flagincorrectnumber);
    unsigned int incorrect_num_blocks=CODESIZE/(incorrect_num_threads)+1;
    
    unsigned int swap_num_threads=min(32,player.flagswapnumber);
    unsigned int swap_num_blocks=CODESIZE/(swap_num_threads)+1;
    //std::cout<<"flagincorrectnumber="<<player.flagincorrectnumber<<std::endl;
    player.flagincorrect=new unsigned int[player.flagincorrectnumber];
    player.flagswap=new unsigned int[player.flagswapnumber];
    unsigned int *device_incorrect_indices;
    unsigned int *device_swap_indices;
    device_incorrect_indices=allocate_device_memory<unsigned int>(CODESIZE);
    device_swap_indices=allocate_device_memory<unsigned int>(CODESIZE);
    
    place_incorrect_swap_values<<<incorrect_num_blocks,incorrect_num_threads>>>(incorrectcheck,device_incorrect_array,device_incorrect_indices,device_incorrect_number,CODESIZE);
    //printdevicearray<<<1,1>>>(device_incorrect_array,player.flagincorrectnumber);
 
    //std::cout<<"Hold up"<<std::endl<<std::endl;
    place_incorrect_swap_values<<<swap_num_blocks,swap_num_threads>>>(swapcheck,device_swap_array,device_swap_indices,device_swap_number,CODESIZE);   
    //printdevicearray<<<1,1>>>(device_swap_array,player.flagswapnumber);
    std::cout<<"am I here 2?"<<std::endl;
    copy_device_to_output<unsigned int>(device_incorrect_array,player.flagincorrect,player.flagincorrectnumber);
    std::cout<<"am I here3?"<<std::endl;
    copy_device_to_output<unsigned int>(device_swap_array,player.flagswap,player.flagswapnumber);
    
    hipFree(incorrectcheck);
    hipFree(swapcheck);
    hipFree(device_incorrect_indices);
    hipFree(device_swap_indices);
    hipFree(device_incorrect_number);
    hipFree(device_incorrect_array);
    hipFree(device_swap_number);
    hipFree(device_swap_array);
    hipFree(devicecode);



}


__global__ void printdevicearray(unsigned int *array,unsigned int number){
    
    for (int i=0;i<number;i++){
        printf("device incorrect[%d]=%u\n",i,array[i]);
    }
}

/*What do I need to do?
First I need to make sure that we compare the playercode to the constcode
FOr every incorrect value, we need to store the key so that we can then copy the indices to modify*/
/*Another function will be used to swap indices
I can then output the number of elements that has incorrect (non-zero) values */
__global__ void gpudevicecheckswapincorrect(unsigned int *devicecode, unsigned int *constcode, int *incorrectcheck, int *swapcheck, unsigned int *incorrectnumber,unsigned int *swapnumber,int numelements){
    int blockId=blockIdx.z*(gridDim.x*gridDim.y)+blockIdx.y*(gridDim.x)+blockIdx.x;
    int index=blockId*(blockDim.x*blockDim.y*blockDim.z)+threadIdx.z*(blockDim.x*blockDim.y)+threadIdx.y*(blockDim.y)+threadIdx.x;

    //__device__ unsigned int incorrectnumbers;
    if(index<numelements){
        gpucheckincorrect(index,devicecode,constcode,incorrectcheck);
        gpucheckswap(index,devicecode,constcode,swapcheck); 
        /*Now we compare whether swap is good*/ 
        
        reduceincorrectonswap(index,incorrectcheck,swapcheck);
        get_key_number_values_add(incorrectcheck[index],incorrectnumber);
        get_key_number_values_add(swapcheck[index],swapnumber);
        /*We now count the number of incorrect elements swap*/

    }
   
    __syncthreads;
    /*Set up the incorrect indices and device_incorrect_swap_array*/

    //*incorrectnumber=incorrectnumbers;
    
}

inline __device__ void gpucheckincorrect(int index, unsigned int *devicecode, unsigned int *constcode, int *incorrectcheck){
    
    incorrectcheck[index]=(devicecode[index]!=constcode[index])*index+(devicecode[index]==constcode[index])*-1;

}

__device__ void gpucheckswap(int index, unsigned int *devicecode, unsigned int *constcode, int *swapcheck){
    
    swapcheck[index]=0;
    int temp=0; 
    int pasttemp=0;
    for (int i=0;i<CODESIZE;i++){
        temp=pasttemp+(devicecode[index]==constcode[i])*(index+1);
        pasttemp=temp;
    }
    temp=pasttemp -1 ;
    swapcheck[index]=temp;
     //printf("hell %d",deviceswapcheck[index]);
    
    __syncthreads;
}

inline __device__ void reduceincorrectonswap(int index, int *incorrectcheck, int *swapcheck){
    unsigned int temp=(swapcheck[index]>=0)*-1+(swapcheck[index]<0)*incorrectcheck[index];
    incorrectcheck[index]=temp;
}

inline __device__ void get_key_number_values_add(int arrayvalue, unsigned int *atomicvalue){
    if (arrayvalue != -1){
        unsigned int val=1;
        atomicAdd(atomicvalue,val);
    }
}


__global__ void place_incorrect_swap_values(int *devicecheck,unsigned int *device_incorrect_swap_array, unsigned int *device_incorrect_swap_indices, unsigned int *device_incorrect_swap_number,unsigned int numelements){
    int blockId=blockIdx.z*(gridDim.x*gridDim.y)+blockIdx.y*(gridDim.x)+blockIdx.x;
    int index=blockId*(blockDim.x*blockDim.y*blockDim.z)+threadIdx.z*(blockDim.x*blockDim.y)+threadIdx.y*(blockDim.y)+threadIdx.x;
    /*I need to make sure the number of threads does not exceed the number of elements*/
    unsigned int range=numelements/(*device_incorrect_swap_number)+1;

    unsigned int start=index*range;
    if (start<numelements){
        unsigned int temp; 
        unsigned int end=(range*(1+index)<numelements)*(range*(1+index))+(range*(1+index)>=numelements)*numelements;
        device_incorrect_swap_indices[index]=0;
        for (int i=start;i<end;i++){
            temp=device_incorrect_swap_indices[index]+(devicecheck[i]>=0); //increment value by 1 if true
            device_incorrect_swap_indices[index]=temp;
        }

        __syncthreads;
        unsigned int newindex=0;
        for (int i=0;i<index;i++){
            temp=newindex+device_incorrect_swap_indices[i];
            newindex=temp;
        }
        unsigned int j=start; //start is a different location in the devicecheck
        for (int i=newindex;i<newindex+device_incorrect_swap_indices[index];i++){
            //device_incorrect_swap_array[i]=1;
            while (devicecheck[j]<0) j++   ;
            device_incorrect_swap_array[i]=(unsigned int)devicecheck[j];
            j++;
            //printf("device_incorrect_swap_array[%d]=%u\n",i,device_incorrect_swap_array[i]);
        }
        
    }
    
    __syncthreads;
        
}

    //I need to incremenent by the number of threads which should equal the number of incorrect array.
    



/* TO PLACE IN A HOST FUNCTION
    //const __global__ unsigned int incorrectnumbers=*incorrectswapnumber;
    /*Set up the incorrect indices and device_incorrect_swap_array
    __global__ unsigned int swap_number_per_thread[incorrectnumbers];
    __global__ unsigned int device_swap[incorrectnumbers];
    if (index<*incorrectswapnumber){
        place_incorrect_swap_values(index,deviceswapcheck,device_swap,swap_number_per_thread,incorrectswapnumber,numelements);
    }
    final_incorrect_array=device_swap;
}
*/ 

__host__ void getcodeattempt(player &player){
/*This will generate a code attempt...Ideally this is done on the CPU stream whilst the other player do their attempt*/
    std::cout<<"another test"<<std::endl;
    if (player.flagincorrectnumber ==0 && player.flagswapnumber ==0){
        for (int i=0;i<CODESIZE;i++){
            
            int key=rand()%player.unused_values.size();
            player.currentcodeattempt[i]=player.unused_values[key];
            player.unused_values.erase(player.unused_values.begin()+key);
        }

    }

    else {
        /*I need to check the swap array*/
        int tempval;
        for (int i=0;i<player.flagswapnumber;i++){
            if (i==0){
                tempval=player.flagswap[i];
            }
            else if (i==player.flagswapnumber-1){
                int index=player.flagswap[i];
                int nextindex=tempval; 
                player.currentcodeattempt[index]=player.currentcodeattempt[nextindex];
            }
            else{
                int index=player.flagswap[i];
                int nextindex=player.flagswap[i+1];
                player.currentcodeattempt[index]=player.currentcodeattempt[nextindex];
            }

            player.currentcodeattempt[i];
        }
    /*Then I need to replace the incorrect values*/
        for (int i=player.flagincorrectnumber;i>=0;i++){
            int index=player.flagincorrect[i];
            int key=rand()%player.unused_values.size();
            player.currentcodeattempt[index]=player.unused_values[key];
            player.unused_values.erase(player.unused_values.begin()+key);   
        }

    }
}
