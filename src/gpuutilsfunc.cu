#include "hip/hip_runtime.h"
/*This file provides all the information data into constant, pinned memory and GPU memory*/
#include <iostream>
#include "common.cuh"
__constant__  unsigned int codetocrack1[CODESIZE]; //2 is for 2 players
__constant__  unsigned int codetocrack2[CODESIZE];

__host__ void copy_to_constant_gpu_memory(unsigned int *hostcodedevice,unsigned int *devicememory,size_t offset){
    /* a reminder that the constant memory is read only and 64kB wide.
     we may need to make sure whether we should set a function to make sure the total 
     codesize does not exceed this number for performance purposes */
     size_t size=CODESIZE*sizeof(unsigned int);
     hipMemcpyToSymbol((void**)&devicememory,(void**)&hostcodedevice,size,0,hipMemcpyHostToDevice);
    //We must make sure that if we end up with one GPU only, the offset gets propagated
}

__host__ void copy_input_to_device(unsigned int *hostdata, unsigned int *devicedata, unsigned int numelements){
    hipError_t error;
    size_t size=numelements*sizeof(unsigned int);
    error=hipMemcpy(devicedata,hostdata,size,hipMemcpyHostToDevice);

}

__host__ void copy_device_to_output(unsigned int *devicedata, unsigned int *hostdata, unsigned int numelements){
    hipError_t error;
    size_t size=numelements*sizeof(unsigned int);
    error=hipMemcpy(hostdata,devicedata,size,hipMemcpyDeviceToHost);
}

__host__ unsigned int* allocate_device_memory(unsigned int numelements){
    hipError_t error;
    unsigned int *devicedata;
    size_t size=numelements*sizeof(unsigned int);
    error=hipMalloc((void**)&devicedata,size);

    return devicedata;
}

__host__ void codecheck(unsigned int *playercode,unsigned int *constcode){
    unsigned int blockspergrid=CODESIZE/THREADSPERBLOCK+1;
    unsigned int *devicecheck; //pointer to device hot encoding must return the index of invalid
    unsigned int *devicecode;
    unsigned int host_incorrect_numbers=0;
    unsigned int *device_incorrect_numbers;
    unsigned int *device_incorrect_array;
    
    devicecheck=allocate_device_memory(CODESIZE);
    devicecode=allocate_device_memory(CODESIZE);
    device_incorrect_numbers=allocate_device_memory(sizeof(unsigned int));
    copy_input_to_device(playercode,devicecode,CODESIZE);
    copy_input_to_device(&host_incorrect_numbers,device_incorrect_numbers,CODESIZE,sizeof(unsigned int));
    /*Now we need to create a new array tp store non zero values*/
    gpu_device_check<<<blockspergrid,THREADSPERBLOCK>>>(devicecode,constcode,devicecheck,incorrectnumbers,CODESIZE);
    copy_device_to_output(device_incorrect_numbers,&host_incorrect_numbers,sizeof(int));
    device_incorrect_array=allocate_device_memory(host_incorrect_numbers);
    //I somehow need to push the zeros to the side so I can later use to resuce the array


    
}

/*What do I need to do?
First I need to make sure that we compare the playercode to the constcode
FOr every incorrect value, we need to store the key so that we can then copy the indices to modify*/
/*Another function will be used to swap indices
I can then output the number of elements that has incorrect (non-zero) values */
__global__ void gpudevicecheck(unsigned int *devicecode, unsigned int *constcode, unsigned int *devicecheck, unsigned int *incorrectnumber,int numelements){
    int blockId=blockIdx.z*(gridDim.x*gridDim.y)+blockIdx.y*(gridDim.x)+blockIdx.x;
    int index=blockId*(blockDim.x*blockDim.y*blockDim.z)+threadIdx.z*(blockDim.x*blockDim.y)+threadIdx.y*(blockDim.y)+threadIdx.x;

    __shared__ unsigned int incorrectnumbers;
    if (index<numelements){
        devicecheck[index]=(devicecode[index]!=constcode[index])*index;
    }
    get_number_of_incorrect_values(devicecheck,index,&incorrectnumbers);
    

}
__device__ void get_number_of_incorrect_values(unsigned int *devicecheck, int index,unsigned int *incorrectvalue){
    if (devicecheck[index] != 0){
        atomicAdd(&incorrectvalue,1);
    }
}

__global__ void cusortarraybubble(){

}
