#include "hip/hip_runtime.h"
/*This file provides all the information data into constant, pinned memory and GPU memory*/
#include <iostream>
#include "common.cuh"
__constant__  unsigned int codetocrack1[CODESIZE]; //2 is for 2 players
__constant__  unsigned int codetocrack2[CODESIZE];

__host__ void copy_to_constant_gpu_memory(unsigned int *hostcodedevice,unsigned int *devicememory,size_t offset){
    /* a reminder that the constant memory is read only and 64kB wide.
     we may need to make sure whether we should set a function to make sure the total 
     codesize does not exceed this number for performance purposes */
     size_t size=CODESIZE*sizeof(unsigned int);
     hipMemcpyToSymbol((void**)&devicememory,(void**)&hostcodedevice,size,0,hipMemcpyHostToDevice);
    //We must make sure that if we end up with one GPU only, the offset gets propagated
}

__host__ void copy_input_to_device(unsigned int *hostdata, unsigned int *devicedata, unsigned int numelements){
    hipError_t error;
    size_t size=numelements*sizeof(unsigned int);
    error=hipMemcpy(devicedata,hostdata,size,hipMemcpyHostToDevice);

}

__host__ void copy_device_to_output(unsigned int *devicedata, unsigned int *hostdata, unsigned int numelements){
    hipError_t error;
    size_t size=numelements*sizeof(unsigned int);
    error=hipMemcpy(hostdata,devicedata,size,hipMemcpyDeviceToHost);
}

__host__ unsigned int* allocate_device_memory(unsigned int numelements){
    hipError_t error;
    unsigned int *devicedata;
    size_t size=numelements*sizeof(unsigned int);
    error=hipMalloc((void**)&devicedata,size);

    return devicedata;
}

__host__ void codecheck(unsigned int *playercode,unsigned int *constcode){
    unsigned int blockspergrid=CODESIZE/THREADSPERBLOCK+1;
    unsigned int *devicecheck; //pointer to device hot encoding must return the index of invalid
    unsigned int *devicecode;
    unsigned int host_incorrect_numbers=0;
    unsigned int *device_incorrect_numbers;
    unsigned int *device_incorrect_array;
    
    devicecheck=allocate_device_memory(CODESIZE);
    devicecode=allocate_device_memory(CODESIZE);
    device_incorrect_numbers=allocate_device_memory(sizeof(unsigned int));
    copy_input_to_device(playercode,devicecode,CODESIZE);
    copy_input_to_device(&host_incorrect_numbers,device_incorrect_numbers,CODESIZE,sizeof(unsigned int));
    /*Now we need to create a new array tp store non zero values*/
    gpu_device_check<<<blockspergrid,THREADSPERBLOCK>>>(devicecode,constcode,devicecheck,incorrectnumbers,CODESIZE);
    copy_device_to_output(device_incorrect_numbers,&host_incorrect_numbers,sizeof(int));
    device_incorrect_array=allocate_device_memory(host_incorrect_numbers);
    //I somehow need to push the zeros to the side so I can later use to resuce the array
    incorrect_num_threads=max(host_incorrect_numbers,32); //32 being the adequate number of threads

    
}

/*What do I need to do?
First I need to make sure that we compare the playercode to the constcode
FOr every incorrect value, we need to store the key so that we can then copy the indices to modify*/
/*Another function will be used to swap indices
I can then output the number of elements that has incorrect (non-zero) values */
__global__ void gpudevicecheckincorrect(unsigned int *devicecode, unsigned int *constcode, unsigned int *devicecheck, unsigned int *incorrectnumber,int numelements){
    int blockId=blockIdx.z*(gridDim.x*gridDim.y)+blockIdx.y*(gridDim.x)+blockIdx.x;
    int index=blockId*(blockDim.x*blockDim.y*blockDim.z)+threadIdx.z*(blockDim.x*blockDim.y)+threadIdx.y*(blockDim.y)+threadIdx.x;

    __shared__ unsigned int incorrectnumbers;
    if (index<numelements){
        devicecheck[index]=(devicecode[index]!=constcode[index])*index;
        get_key_number_values(devicecheck,index,&incorrectnumbers);
    }
    
    

}
__device__ void get_key_number_values(unsigned int *devicecheck, int index,unsigned int *incorrectvalue){
    if (devicecheck[index] != 0){
        atomicAdd(&incorrectvalue,1);
    }
}

__global__ void place_incorrect_swap_values(unsigned int *devicecheck,unsigned int *device_incorrect_array, int numelements, unsigned int numthreads){
    int blockId=blockIdx.z*(gridDim.x*gridDim.y)+blockIdx.y*(gridDim.x)+blockIdx.x;
    int startindex=blockId*(blockDim.x*blockDim.y*blockDim.z)+threadIdx.z*(blockDim.x*blockDim.y)+threadIdx.y*(blockDim.y)+threadIdx.x;
    int index=startindex;
    //I need to incremenent by the number of threads which should equal the number of incorrect array.
    while (devicecheck[index]=0 && index < numelements){
         index +=numthreads; //increment by the number of threads
    }
    device_incorrect_array[startindex]=devicecheck[index];
    __syncthreads;
}

__global__ void gpudevicecheckincorrect(unsigned int *devicecode, unsigned int *constcode, unsigned int *deviceswapcheck, unsigned int *incorrectswapnumber,int numelements){
    int blockId=blockIdx.z*(gridDim.x*gridDim.y)+blockIdx.y*(gridDim.x)+blockIdx.x;
    int index=blockId*(blockDim.x*blockDim.y*blockDim.z)+threadIdx.z*(blockDim.x*blockDim.y)+threadIdx.y*(blockDim.y)+threadIdx.x;

    if (index<numelements){
        unsigned int temp=0; 
        unsigned int pasttemp=0;
        for (int i=0){
            temp=pasttemp+(devicecode[index]==constcode[i])*index;
            pasttemp=temp;
        }
         deviceswapcheck[index]=temp;
        get_key_number_values(deviceswapcheck,index,&incorrectswapnumbers);
    }

}

