#include "hip/hip_runtime.h"
/*This file provides all the information data into constant, pinned memory and GPU memory*/
#include <iostream>
#include "common.cuh"
#include "gpuutilsfunc.cuh"
#include <string.h>
__constant__  unsigned int codetocrack1[CODESIZE]; //2 is for 2 players
__constant__  unsigned int codetocrack2[CODESIZE];

__global__ void testval(unsigned int *constantmemory1, unsigned int *constantmemory2,unsigned int *deviceval){
    int blockId=blockIdx.z*(gridDim.x*gridDim.y)+blockIdx.y*(gridDim.x)+blockIdx.x;
    int index=blockId*(blockDim.x*blockDim.y*blockDim.z)+threadIdx.z*(blockDim.x*blockDim.y)+threadIdx.y*(blockDim.y)+threadIdx.x;
    if (index==0) *deviceval=constantmemory1[0]+constantmemory2[2];
    
        
    }
__host__  void copy_to_constant_gpu_memory(player &player,int playernumber){
    /* a reminder that the constant memory is read only and 64kB wide.
     we may need to make sure whether we should set a function to make sure the total 
     codesize does not exceed this number for performance purposes */
     size_t size=CODESIZE*sizeof(unsigned int);
     if (playernumber==0){
        hipError_t err=hipMemcpyToSymbol(HIP_SYMBOL(codetocrack1),player.hostcode,size);
        hipGetSymbolAddress((void**)&player.constantmemory,codetocrack1);
     }

     else{
        hipError_t err=hipMemcpyToSymbol(HIP_SYMBOL(codetocrack2),player.hostcode,size);
        hipGetSymbolAddress((void**)&player.constantmemory,codetocrack2);
     }
    
        /*if(strcmp(deviceconstantmemory,"crack2")==0){
            std::cout<<"test"<<std::endl;
            hipError_t err=hipMemcpyToSymbol(HIP_SYMBOL(codetocrack1),hostcodedevice,size);
            if (err!=hipSuccess){
                printf("%s",hipGetErrorString(err));
                exit(1);
        }
        
        }*/
       
        std::cout<<"test"<<std::endl;
    //We must make sure that if we end up with one GPU only, the offset gets propagated
}
template <typename T>
__host__ void copy_input_to_device(T *hostdata, T *devicedata, T numelements){
    hipError_t error;
    size_t size=numelements*sizeof(T);
    error=hipMemcpy(devicedata,hostdata,size,hipMemcpyHostToDevice);

}
template <typename T>
__host__ void copy_device_to_output(T *devicedata, T *hostdata, T numelements){
    hipError_t error;
    size_t size=numelements*sizeof(T);
    error=hipMemcpy(hostdata,devicedata,size,hipMemcpyDeviceToHost);
}

__host__ unsigned int* allocate_device_memory(unsigned int numelements){
    hipError_t error;
    unsigned int *devicedata;
    size_t size=numelements*sizeof(unsigned int);
    error=hipMalloc((void**)&devicedata,size);

    return devicedata;
}


__host__ void codecheck(player &player){
    get_incorrect_array(player);
    get_swap_array(player);


    /* I need to free all pointers and CUDA variables*/
    
}


__host__ void get_incorrect_array(player &player){
    unsigned int blockspergrid=CODESIZE/THREADSPERBLOCK+1;
    unsigned int *devicecode;
    unsigned int *device_incorrect_number;
    unsigned int *device_incorrect_array;
    unsigned int *devicecheck;
    unsigned int host_incorrect_number=0;
    
    devicecheck=allocate_device_memory(CODESIZE);
    devicecode=allocate_device_memory(CODESIZE);
    device_incorrect_number=allocate_device_memory(sizeof(unsigned int));
    copy_input_to_device<unsigned int>(player.currentcodeattempt,devicecode,CODESIZE);
    copy_input_to_device<unsigned int>(&player.flagincorrectnumber,device_incorrect_number,1);
    
    gpudevicecheckincorrect<<<blockspergrid,THREADSPERBLOCK>>>(devicecode,player.constantmemory,devicecheck,device_incorrect_number,CODESIZE);
    copy_device_to_output<unsigned int>(device_incorrect_number,&player.flagincorrectnumber,1);
    device_incorrect_array=allocate_device_memory(player.flagincorrectnumber);

    int incorrect_num_threads=max(player.flagincorrectnumber,32);
    int incorrect_num_blocks=player.flagincorrectnumber/incorrect_num_threads + 1 ; 
    int total_num_threads=incorrect_num_threads*incorrect_num_blocks;
    place_incorrect_swap_values<<<incorrect_num_blocks,incorrect_num_threads>>>(devicecheck,device_incorrect_array, device_incorrect_number, total_num_threads);
    /*Place the data back into the player values*/
    /*I will need to free these values before reinitialize them*/
    copy_device_to_output<unsigned int>(device_incorrect_number,&player.flagincorrectnumber,1);
    player.flagincorrect=new unsigned int[player.flagincorrectnumber];
      
    copy_device_to_output<unsigned int>(device_incorrect_array,player.flagincorrect,player.flagincorrectnumber);    
    hipFree(devicecheck);
    hipFree(device_incorrect_number);
    hipFree(device_incorrect_array);
    hipFree(devicecode);



}

__host__ void get_swap_array(player &player){
    unsigned int blockspergrid=CODESIZE/THREADSPERBLOCK+1;
    unsigned int *devicecode;
    unsigned int *devicecheck;
    unsigned int host_swapable_number=0;
    unsigned int *device_swapable_number;
    unsigned int *device_swapable_array;
     
    devicecheck=allocate_device_memory(CODESIZE);
    devicecode=allocate_device_memory(CODESIZE);
    device_swapable_number=allocate_device_memory(sizeof(unsigned int));
    copy_input_to_device<unsigned int>(player.currentcodeattempt,devicecode,CODESIZE);
    copy_input_to_device<unsigned int>(&player.flagswapnumber,device_swapable_number,1);

    gpudevicecheckswap<<<blockspergrid,THREADSPERBLOCK>>>(devicecode,player.constantmemory,devicecheck,device_swapable_number,CODESIZE);
    copy_device_to_output<unsigned int>(device_swapable_number,&player.flagswapnumber,1);    
    device_swapable_array=allocate_device_memory(player.flagswapnumber);
    
    int swap_num_threads=max(player.flagswapnumber,32); //32 being the adequate number of threads
    int swap_num_blocks=player.flagswapnumber/swap_num_threads + 1 ; 
    std::cout<<"num blocks: "<< swap_num_blocks<<std::endl;
    int total_num_threads=swap_num_threads*swap_num_blocks;


    place_incorrect_swap_values<<<swap_num_blocks,swap_num_threads>>>(devicecheck,device_swapable_array,device_swapable_number,total_num_threads);
    player.flagswap=new unsigned int[player.flagswapnumber];
    copy_device_to_output<unsigned int>(device_swapable_array,player.flagswap,player.flagswapnumber);

    hipFree(devicecode);
    hipFree(devicecheck);
    hipFree(device_swapable_number);
    hipFree(device_swapable_array);
}

/*What do I need to do?
First I need to make sure that we compare the playercode to the constcode
FOr every incorrect value, we need to store the key so that we can then copy the indices to modify*/
/*Another function will be used to swap indices
I can then output the number of elements that has incorrect (non-zero) values */
__global__ void gpudevicecheckincorrect(unsigned int *devicecode, unsigned int *constcode, unsigned int *devicecheck, unsigned int *incorrectnumber,int numelements){
    int blockId=blockIdx.z*(gridDim.x*gridDim.y)+blockIdx.y*(gridDim.x)+blockIdx.x;
    int index=blockId*(blockDim.x*blockDim.y*blockDim.z)+threadIdx.z*(blockDim.x*blockDim.y)+threadIdx.y*(blockDim.y)+threadIdx.x;

    __global__ unsigned int incorrectnumbers;
    //printf("%u ",devicecheck[index]);
    if (index<numelements){
        devicecheck[index]=(devicecode[index]!=constcode[index])*index;
        
        get_key_number_values(devicecheck,index,incorrectnumber);
    }
    __syncthreads;
    //*incorrectnumber=incorrectnumbers;
    
}
__device__ void get_key_number_values(unsigned int *devicecheck, int index,unsigned int *incorrectvalue){
    if (devicecheck[index] != 0){
        unsigned int val=1;
        atomicAdd(incorrectvalue,val);
    }
}

__global__ void place_incorrect_swap_values(unsigned int *devicecheck,unsigned int *device_incorrect_array, unsigned int *numelements,int numthreads){
    int blockId=blockIdx.z*(gridDim.x*gridDim.y)+blockIdx.y*(gridDim.x)+blockIdx.x;
    int startindex=blockId*(blockDim.x*blockDim.y*blockDim.z)+threadIdx.z*(blockDim.x*blockDim.y)+threadIdx.y*(blockDim.y)+threadIdx.x;
    unsigned int index=startindex;
    //I need to incremenent by the number of threads which should equal the number of incorrect array.
    while (devicecheck[index]==0 && startindex < *numelements && index < CODESIZE){
         index +=numthreads; //increment by the number of threads
         //printf("%u ",devicecheck[index]);
    }
    device_incorrect_array[startindex]=devicecheck[index];
    __syncthreads;
}

__global__ void gpudevicecheckswap(unsigned int *devicecode, unsigned int *constcode, unsigned int *deviceswapcheck, unsigned int *incorrectswapnumber,int numelements){
    int blockId=blockIdx.z*(gridDim.x*gridDim.y)+blockIdx.y*(gridDim.x)+blockIdx.x;
    int index=blockId*(blockDim.x*blockDim.y*blockDim.z)+threadIdx.z*(blockDim.x*blockDim.y)+threadIdx.y*(blockDim.y)+threadIdx.x;

    if (index<numelements){
        unsigned int temp=0; 
        unsigned int pasttemp=0;
        for (int i=0;i<CODESIZE;i++){
            temp=pasttemp+(devicecode[index]==constcode[i])*index;
            pasttemp=temp;
        }
         deviceswapcheck[index]=temp;
        get_key_number_values(deviceswapcheck,index,incorrectswapnumber);
    }

}


__host__ void getcodeattempt(player &player){
/*This will generate a code attempt...Ideally this is done on the CPU stream whilst the other player do their attempt*/
    std::cout<<"another test"<<std::endl;
    if (player.flagincorrectnumber ==0 && player.flagswapnumber ==0){
        for (int i=0;i<CODESIZE;i++){
            
            int key=rand()%player.unused_values.size();
            player.currentcodeattempt[i]=player.unused_values[key];
            player.unused_values.erase(player.unused_values.begin()+key);
        }

    }

    else {
        /*I need to check the swap array*/
        int tempval;
        for (int i=0;i<player.flagswapnumber;i++){
            if (i==0){
                tempval=player.flagswap[i];
            }
            else if (i==player.flagswapnumber-1){
                int index=player.flagswap[i];
                int nextindex=tempval; 
                player.currentcodeattempt[index]=player.currentcodeattempt[nextindex];
            }
            else{
                int index=player.flagswap[i];
                int nextindex=player.flagswap[i+1];
                player.currentcodeattempt[index]=player.currentcodeattempt[nextindex];
            }

            player.currentcodeattempt[i];
        }
    /*Then I need to replace the incorrect values*/
        for (int i=player.flagincorrectnumber;i>=0;i++){
            int index=player.flagincorrect[i];
            int key=rand()%player.unused_values.size();
            player.currentcodeattempt[index]=player.unused_values[key];
            player.unused_values.erase(player.unused_values.begin()+key);   
        }

    }
}
