#include "hip/hip_runtime.h"
/*This file provides all the information data into constant, pinned memory and GPU memory*/
#include <iostream>
#include "common.cuh"
#include "gpuutilsfunc.cuh"
__constant__  unsigned int codetocrack1[CODESIZE]; //2 is for 2 players
__constant__  unsigned int codetocrack2[CODESIZE];

__host__ void copy_to_constant_gpu_memory(unsigned int *hostcodedevice,unsigned int *deviceconstantmemory,size_t offset){
    /* a reminder that the constant memory is read only and 64kB wide.
     we may need to make sure whether we should set a function to make sure the total 
     codesize does not exceed this number for performance purposes */
     size_t size=CODESIZE*sizeof(unsigned int);
     hipMemcpyToSymbol((void**)&deviceconstantmemory,(void**)&hostcodedevice,size,0,hipMemcpyHostToDevice);
    //We must make sure that if we end up with one GPU only, the offset gets propagated
}

__host__ void copy_input_to_device(unsigned int *hostdata, unsigned int *devicedata, unsigned int numelements){
    hipError_t error;
    size_t size=numelements*sizeof(unsigned int);
    error=hipMemcpy(devicedata,hostdata,size,hipMemcpyHostToDevice);

}

__host__ void copy_device_to_output(unsigned int *devicedata, unsigned int *hostdata, unsigned int numelements){
    hipError_t error;
    size_t size=numelements*sizeof(unsigned int);
    error=hipMemcpy(hostdata,devicedata,size,hipMemcpyDeviceToHost);
}

__host__ unsigned int* allocate_device_memory(unsigned int numelements){
    hipError_t error;
    unsigned int *devicedata;
    size_t size=numelements*sizeof(unsigned int);
    error=hipMalloc((void**)&devicedata,size);

    return devicedata;
}


__host__ void codecheck(player &player,unsigned int *constcode){
    get_incorrect_array(player &player, unsigned int *costcode);
    get_swap_array(player &player, unsigned int *costcode);


    /* I need to free all pointers and CUDA variables*/
    
}


__host__ void get_incorrect_array(player &player,unsigned int *constcode){
    unsigned int blockspergrid=CODESIZE/THREADSPERBLOCK+1;
    unsigned int *devicecode;
    unsigned int *device_incorrect_number;
    unsigned int *device_incorrect_array;
    unsigned int *devicecheck;
    unsigned int host_incorrect_number=0;
    
    devicecheck=allocate_device_memory(CODESIZE);
    devicecode=allocate_device_memory(CODESIZE);
    device_incorrect_number=allocate_device_memory(sizeof(unsigned int));
    copy_input_to_device(player->currentcodeattempt,devicecode,CODESIZE);
    copy_input_to_device(&host_incorrect_number,device_incorrect_number,sizeof(unsigned int));
    
    gpudevicecheckincorrect<<<blockspergrid,THREADSPERBLOCK>>>(devicecode,constcode,devicecheck,device_incorrect_number,CODESIZE);
    copy_device_to_output(device_incorrect_number,&host_incorrect_number,sizeof(unsigned int));
    device_incorrect_array=allocate_device_memory(host_incorrect_number);

    int incorrect_num_threads=max(host_incorrect_number,32);
    int incorrect_num_blocks=host_incorrect_number/incorrect_num_threads + 1 ; 

    place_incorrect_swap_value<<<incorrect_num_blocks,incorrect_num_threads>>>(devicecheck,device_incorrect_array, device_incorrect_number);
    /*Place the data back into the player values*/
    /*I will need to free these values before reinitialize them*/
    player.flagincorrect=new unsigned int[host_incorrect_numbers];
    copy_device_to_output(device_incorrect_number,&host_incorrect_number,sizeof(unsigned int));
    
    copy_device_to_output(device_incorrect_array,player.flagincorrect,host_incorrect_numbers*sizeof(unsigned int));    
    cudafree(devicecheck);
    cudafree(device_incorrect_number);
    cudafree(device_incorrect_array);
    cudafree(devicecode);



}

__host__ void get_swap_array(player &player,unsigned int *constcode){
    unsigned int blockspergrid=CODESIZE/THREADSPERBLOCK+1;
    unsigned int *devicecode;
    unsigned int *devicecheck;
    unsigned int host_swapable_number=0;
    unsigned int *device_swapable_number;
    unsigned int *device_swapable_array;
     
    devicecheck=allocate_device_memory(CODESIZE);
    devicecode=allocate_device_memory(CODESIZE);
    device_swapable_number=allocate_device_memory(sizeof(unsigned int));
    copy_input_to_device(player->currentcodeattempt,devicecode,CODESIZE);
    copy_input_to_device(&host_incorrect_number,device_incorrect_number,sizeof(unsigned int));
    copy_input_to_device(&host_swapable_number,device_swapable_number,sizeof(unsigned int));

    gpudevicecheckswap<<<blockspergrid,THREADSPERBLOCK>>>(devicecode,constcode,devicecheck,device_swapable_number,CODESIZE);
    copy_device_to_output(device_swapable_number,&host_swapable_number,sizeof(unsigned int));    
    device_swapable_array=allocate_device_memory(host_swapable_number);
    
    int incorrect_num_threads=max(host_incorrect_number,32); //32 being the adequate number of threads
    int incorrect_num_blocks=host_incorrect_number/incorrect_num_threads + 1 ; 

    place_incorrect_swap_value<<<incorrect_num_blocks,incorrect_num_threads>>>(devicecheck,device_swapable_array,device_swapable_number);
    player.flagswap=new unsigned int[host_swapable_number];
    copy_device_to_output(device_swapable_array,player.flagswap,host_swapable_number*sizeof(unsigned int));

    cudafree(devicecode);
    cudafree(devicecheck);
    cudafree(device_swapable_number);
    cudafree(device_swapable_array);
}

/*What do I need to do?
First I need to make sure that we compare the playercode to the constcode
FOr every incorrect value, we need to store the key so that we can then copy the indices to modify*/
/*Another function will be used to swap indices
I can then output the number of elements that has incorrect (non-zero) values */
__global__ void gpudevicecheckincorrect(unsigned int *devicecode, unsigned int *constcode, unsigned int *devicecheck, unsigned int *incorrectnumber,int numelements){
    int blockId=blockIdx.z*(gridDim.x*gridDim.y)+blockIdx.y*(gridDim.x)+blockIdx.x;
    int index=blockId*(blockDim.x*blockDim.y*blockDim.z)+threadIdx.z*(blockDim.x*blockDim.y)+threadIdx.y*(blockDim.y)+threadIdx.x;

    __shared__ unsigned int incorrectnumbers;
    if (index<numelements){
        devicecheck[index]=(devicecode[index]!=constcode[index])*index;
        get_key_number_values(devicecheck,index,&incorrectnumbers);
    }
    
    

}
__device__ void get_key_number_values(unsigned int *devicecheck, int index,unsigned int *incorrectvalue){
    if (devicecheck[index] != 0){
        atomicAdd(&incorrectvalue,1);
    }
}

__global__ void place_incorrect_swap_values(unsigned int *devicecheck,unsigned int *device_incorrect_array, int numelements){
    int blockId=blockIdx.z*(gridDim.x*gridDim.y)+blockIdx.y*(gridDim.x)+blockIdx.x;
    int startindex=blockId*(blockDim.x*blockDim.y*blockDim.z)+threadIdx.z*(blockDim.x*blockDim.y)+threadIdx.y*(blockDim.y)+threadIdx.x;
    int index=startindex;
    //I need to incremenent by the number of threads which should equal the number of incorrect array.
    while (devicecheck[index]=0 && index < numelements){
         index +=numthreads; //increment by the number of threads
    }
    device_incorrect_array[startindex]=devicecheck[index];
    __syncthreads;
}

__global__ void gpudevicecheckswap(unsigned int *devicecode, unsigned int *constcode, unsigned int *deviceswapcheck, unsigned int *incorrectswapnumber,int numelements){
    int blockId=blockIdx.z*(gridDim.x*gridDim.y)+blockIdx.y*(gridDim.x)+blockIdx.x;
    int index=blockId*(blockDim.x*blockDim.y*blockDim.z)+threadIdx.z*(blockDim.x*blockDim.y)+threadIdx.y*(blockDim.y)+threadIdx.x;

    if (index<numelements){
        unsigned int temp=0; 
        unsigned int pasttemp=0;
        for (int i=0){
            temp=pasttemp+(devicecode[index]==constcode[i])*index;
            pasttemp=temp;
        }
         deviceswapcheck[index]=temp;
        get_key_number_values(deviceswapcheck,index,&incorrectswapnumbers);
    }

}


__host__ generatecodeattempt(player &player){
/*This will generate a code attempt*/
if (player.flagincorrect ==NULL && player.flagswap ==NULL){

    
}
}
