/*This file provides all the information data into constant, pinned memory and GPU memory*/
#include <iostream>
#include "common.cuh"
__constant__  unsigned int codetocrack1[CODESIZE]; //2 is for 2 players
__constant__  unsigned int codetocrack2[CODESIZE];

__host__ void copy_to_constant_gpu_memory(unsigned int *hostcodedevice,unsigned int *devicememory,size_t offset){
    /* a reminder that the constant memory is read only and 64kB wide.
     we may need to make sure whether we should set a function to make sure the total 
     codesize does not exceed this number for performance purposes */
     size_t size=CODESIZE*sizeof(unsigned int);
     hipMemcpyToSymbol((void**)&devicememory,(void**)&hostcodedevice,size,0,hipMemcpyHostToDevice);
    //We must make sure that if we end up with one GPU only, the offset gets propagated
}

__host__ void copy_input_to_device(unsigned int *hostdata, unsigned int *devicedata, unsigned int numelements){
    hipError_t error;
    size_t size=numelements*sizeof(unsigned int);
    error=hipMemcpy(devicedata,hostdata,size,hipMemcpyHostToDevice);

}

__host__ void copy_device_to_output(unsigned int *devicedata, unsigned int *hostdata, unsigned int numelements){
    hipError_t error;
    size_t size=numelements*sizeof(unsigned int);
    error=hipMemcpy(hostdata,devicedata,size,hipMemcpyDeviceToHost);
}

__host__ unsigned int* allocate_device_memory(unsigned int numelements){
    hipError_t error;
    unsigned int *devicedata;
    size_t size=numelements*sizeof(unsigned int);
    error=hipMalloc((void**)&devicedata,size);

    return devicedata;
}



